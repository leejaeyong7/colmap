#include "hip/hip_runtime.h"
// Copyright (c) 2018, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#include "mvs/gpu_mat_ref_image.h"

#include <iostream>

#include "util/cudacc.h"

namespace colmap {
namespace mvs {
namespace {

// reference image texture is W x H x C float typed texture
texture<float, hipTextureType2DLayered> image_texture;
__global__ void FilterKernel(GpuMat<float> image, GpuMat<float> sum_image,
                             GpuMat<float> squared_sum_image,
                             const int window_radius, const int window_step,
                             const float sigma_spatial,
                             const float sigma_color) {
  const size_t row = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t col = blockDim.x * blockIdx.x + threadIdx.x;
  if (row >= image.GetHeight() || col >= image.GetWidth()) {
    return;
  }
  const int channel = image.GetDepth();
  // MultiChannelWeightComputer multi_channel_weight_computer_(sigma_spatial, sigma_color, channel);
  // const float center_feature[channel];
  for(int i= 0; i < channel; i++){
   const float feat = tex2DLayered(image_texture, col, row, i);
   image.Set(row, col, i, feat);
  }

  // float feature_sum[channel];
  // float feature_squared_sum[channel];
  // float bilateral_weight_sum = 0.0f;

  // for (int window_row = -window_radius; window_row <= window_radius;
  //      window_row += window_step) {
  //   for (int window_col = -window_radius; window_col <= window_radius;
  //        window_col += window_step) {
  //     const float feature[channel];
  //     for(int c = 0; c < channel; c++){
  //       feature[c] = tex2DLayered(image_texture,
  //         col + window_col,
  //         row + window_row, c);
  //     }
  //     const float multi_channel_weight = multi_channel_weight_computer_.Compute(
  //         window_row, window_col, center_feature, feature, channel);
  //     for(auto i = 0; i < channel; i++){
  //       feature_sum[i] += multi_channel_weight * feature[i];
  //       feature_squared_sum[i] += multi_channel_weight * feature[i] * feature[i];
  //     }
  //     bilateral_weight_sum += multi_channel_weight;
  //   }
  // }
  // for(auto i = 0; i < channel; i++){
  //   feature_sum[i] /= multi_channel_weight;
  //   feature_squared_sum[i] /= multi_channel_weight;
  // }

  // image.SetSlice(row, col, center_feature);
  // sum_image.SetSlice(row, col, feature_sum);
  // squared_sum_image.SetSlice(row, col, feature_squared_sum);
}

}  // namespace

// Adding Channel as input in constructor
GpuMatRefImage::GpuMatRefImage(const size_t width, const size_t height,
                               const size_t channel)
    : height_(height), width_(width), channel_(channel){
  image.reset(new GpuMat<float>(width, height, channel));
  sum_image.reset(new GpuMat<float>(width, height, channel));
  squared_sum_image.reset(new GpuMat<float>(width, height, channel));
}

void GpuMatRefImage::Filter(const float* image_data,
                            const size_t window_radius,
                            const size_t window_step, const float sigma_spatial,
                            const float sigma_color) {
  // adding channel as input
  CudaArrayWrapper<float> image_array(width_, height_, channel_);
  image_array.CopyToDevice(image_data);
  image_texture.addressMode[0] = hipAddressModeBorder;
  image_texture.addressMode[1] = hipAddressModeBorder;
  image_texture.addressMode[2] = hipAddressModeBorder;
  image_texture.filterMode = hipFilterModePoint;
  image_texture.normalized = false;

  const dim3 block_size(kBlockDimX, kBlockDimY);
  const dim3 grid_size((width_ - 1) / block_size.x + 1,
                       (height_ - 1) / block_size.y + 1);

  CUDA_SAFE_CALL(hipBindTextureToArray(image_texture, image_array.GetPtr()));
  FilterKernel<<<grid_size, block_size>>>(
      *image, *sum_image, *squared_sum_image, window_radius, window_step,
      sigma_spatial, sigma_color);
  CUDA_SYNC_AND_CHECK();
  CUDA_SAFE_CALL(hipUnbindTexture(image_texture));
}

}  // namespace mvs
}  // namespace colmap
